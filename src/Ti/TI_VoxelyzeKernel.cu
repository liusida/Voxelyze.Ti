#include "hip/hip_runtime.h"
#include <iostream>

#include "TI_VoxelyzeKernel.h"

CTI_VoxelyzeKernel::CTI_VoxelyzeKernel( CVoxelyze* vx )
{
    _vx = vx;
}

CTI_VoxelyzeKernel::~CTI_VoxelyzeKernel()
{
}

__global__ void gpu_function_1(int* a, int num) {
    int gindex = threadIdx.x + blockIdx.x * blockDim.x; 
    if (gindex < num) {
        a[gindex] = gindex;
    }
}

void CTI_VoxelyzeKernel::simpleGPUFunction() {
    int* d_a;
    int* a;
    int num = 10;
    int mem_size = num * sizeof(int);

    a = (int *) malloc(mem_size);
    hipMalloc( &d_a, mem_size );

    gpu_function_1<<<1,num>>>(d_a, num);
    hipMemcpy(a, d_a, mem_size, hipMemcpyDeviceToHost);

    for (int i=0;i<num;i++) {
        std::cout<< a[i] << ",";
    }
    std::cout << std::endl;
}

void CTI_VoxelyzeKernel::doTimeStep(double dt) {

}